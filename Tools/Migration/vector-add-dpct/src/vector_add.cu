//==============================================================
// Copyright � 2019 Intel Corporation
//
// SPDX-License-Identifier: MIT
// =============================================================

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define VECTOR_SIZE 256

__global__ void VectorAddKernel(float* A, float* B, float* C)
{
    A[threadIdx.x] = threadIdx.x + 1.0f;
    B[threadIdx.x] = threadIdx.x + 1.0f;
    C[threadIdx.x] = A[threadIdx.x] + B[threadIdx.x];
}

int main()
{
    float *d_A, *d_B, *d_C;
    hipError_t status;

    hipMalloc(&d_A, VECTOR_SIZE*sizeof(float));
    hipMalloc(&d_B, VECTOR_SIZE*sizeof(float));
    hipMalloc(&d_C, VECTOR_SIZE*sizeof(float));
    
    VectorAddKernel<<<1, VECTOR_SIZE>>>(d_A, d_B, d_C);
    
    float Result[VECTOR_SIZE] = { };
    
    status = hipMemcpy(Result, d_C, VECTOR_SIZE*sizeof(float), hipMemcpyDeviceToHost);
    if (status != hipSuccess) {
        printf("Could not copy result to host\n");
        exit(EXIT_FAILURE);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    for (int i = 0; i < VECTOR_SIZE; i++) {
        if (i % 16 == 0) {
            printf("\n");
        }
        printf("%3.0f ", Result[i]);    
    }
    printf("\n");
	
    return 0;
}
